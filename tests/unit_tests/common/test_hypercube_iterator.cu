#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include "./test_hypercube_iterator.cpp"
#include <gridtools/common/cuda_util.hpp>

static const size_t Size = 2;

GT_FUNCTION int linear_index(gridtools::array<size_t, 2> &index) { return index[0] * Size + index[1]; }

__global__ void test_kernel(int *out_ptr) {
    for (size_t i = 0; i < Size * Size; ++i)
        out_ptr[i] = -1;

    using hypercube_t = gridtools::array<gridtools::array<size_t, 2>, 2>;
    for (auto pos : make_hypercube_view(hypercube_t{{{0ul, Size}, {0ul, Size}}})) {
        out_ptr[linear_index(pos)] = linear_index(pos);
    }
};

TEST(multi_iterator, iterate_on_device) {
    int *out;
    GT_CUDA_CHECK(hipMalloc(&out, sizeof(int) * Size * Size));

    hipLaunchKernelGGL(test_kernel, dim3(1), dim3(1), 0, 0, out);

    int host_out[Size * Size];
    GT_CUDA_CHECK(hipMemcpy(&host_out, out, sizeof(int) * Size * Size, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < Size * Size; ++i)
        ASSERT_EQ(i, host_out[i]) << "at i = " << i;
}
